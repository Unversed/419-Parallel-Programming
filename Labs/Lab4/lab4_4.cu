#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16
#define seed 13

__global__ void matrixMul(float *dev_A, float *dev_B, float *dev_C, int
      matrixWitdh)
{
   __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
   __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

   int bx = blockIdx.x;
   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int row = by * TILE_WIDTH + ty;
   int col = bx * TILE_WIDTH + tx;

   float partial = 0.0;
   int m;
   for( m=0 ; m < matrixWitdh/TILE_WIDTH; m++){
      A_tile[ty][tx] = dev_A[row * matrixWitdh + (m * TILE_WIDTH + tx)];
      B_tile[ty][tx] = dev_B[col + (m * TILE_WIDTH + ty) * matrixWitdh];
      __syncthreads();
      int k;
      for(k=0; k< TILE_WIDTH; k++)
         partial += A_tile[ty][k] * B_tile[k][tx];
      __syncthreads();
      dev_C[row * matrixWitdh + col] = partial;
   }

}

int main(int argc, char **argv){
   srand(seed);

   if(argc != 2){
      printf("Usage /lab4_4 <matrixWitdh>");
      return 1;
   }
   int matrixWitdh = atoi(argv[1]);

   float *h_A = (float*) malloc(matrixWitdh * matrixWitdh * sizeof(float));
   float *h_B = (float*) malloc(matrixWitdh * matrixWitdh * sizeof(float));
   float *h_C = (float*) malloc(matrixWitdh * matrixWitdh * sizeof(float));

   int i,j;
   for(i=0;i<matrixWitdh;i++){
      for(j=0;j<matrixWitdh;j++){
         h_A[i * matrixWitdh + j] = (float)rand()/((float)RAND_MAX/10.0);
         h_B[i * matrixWitdh + j] = (float)rand()/((float)RAND_MAX/10.0);
      }
   }

   float *d_A, *d_B, *d_C;
   hipMalloc((void**) &d_A, matrixWitdh * matrixWitdh * sizeof(float));
   hipMalloc((void**) &d_B, matrixWitdh * matrixWitdh * sizeof(float));
   hipMalloc((void**) &d_C, matrixWitdh * matrixWitdh * sizeof(float));

   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 2);
   dim3 dimGrid(matrixWitdh/TILE_WIDTH, matrixWitdh/TILE_WIDTH, 1);

   float elapsedTime;
   hipEvent_t start, stop;

   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);
   hipMemcpy(d_A, h_A, matrixWitdh * matrixWitdh * sizeof(float),
         hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, matrixWitdh * matrixWitdh * sizeof(float),
         hipMemcpyHostToDevice);
   matrixMul<<< dimGrid, dimBlock >>>(d_A, d_B, d_C, matrixWitdh);
   hipMemcpy(h_C, d_C, matrixWitdh* matrixWitdh * sizeof(float),
         hipMemcpyDeviceToHost);
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start, stop);
   hipEventDestroy(start);
   hipEventDestroy(stop);
   printf("For tiled version, the elapsed time is %.4f(ms).\n", elapsedTime);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
}
